#include "hip/hip_runtime.h"
#include<iostream>
#include<hipblas.h>

int main() {
    // initiliaze the size
    const int N = 1 << 10, bytes = N*N*sizeof(int);
    
    // Allocate managed memory
    float *A, *B, *C;
    hipMallocManaged(&A, bytes);
    hipMallocManaged(&B, bytes);
    hipMallocManaged(&C, bytes);

    // initialize the matrices
    for (int r=0; r<N; ++r) {
        for (int c=0; c<N; ++c) {
            int i = r*N + c;
            A[i] = B[i] = 1;
        }
    }

    // Create CUBLAS handle
    hipblasHandle_t blas;
    hipblasCreate(&blas);

    // Multiply C = (alpha * A) * B + (beta * C)
    float alpha = 1, beta = 0;
    hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A, N, B, N, &beta, C, N);

    // Synchronize with GPU
    hipDeviceSynchronize();

    // Destroy
    hipblasDestroy(blas);

    // Validate the matrix
    float maxError = 0;
    for (int r=0; r<N; ++r) {
        for (int c=0; c<N; ++c) {
            maxError = max(maxError, abs(C[r*N + c] - N));
        }
    }
    std::cout << "Max Error: " << maxError << std::endl;

    // Free the managed memory
    for (int i=0; i<N; ++i) {
        hipFree(A);
        hipFree(B);
        hipFree(C);
    }
    
    return 0;
}